#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vector_add(const int *a, const int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    // mem op starts
    int a, b, c;
    printf("a = "); scanf("%d", &a);
    printf("b = "); scanf("%d", &b);
    // mem op ends

    // gmem op starts
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, sizeof(int));
    hipMalloc((void **)&dev_b, sizeof(int));
    hipMalloc((void **)&dev_c, sizeof(int));
    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);
    vector_add <<<1, 1>>> (dev_a, dev_b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    // gmem op ends

    printf("c = %d + %d = %d\n", a, b, c);
    return 0;
}

